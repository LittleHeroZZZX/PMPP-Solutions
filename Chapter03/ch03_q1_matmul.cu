
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 32

__global__ void matmul_row_kernel(float *M, float *N, float *P, int width){
    /*
     * each thread compute one row of P
     * both block and grid size are 1D
     * i-th row of P involves the i-th row of M and all columns of N
     */
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    if(row < width){
        for(int j=0; j<width; j++){ // j iterates over N's columns
            float sum = 0;
            for(int i=0; i<width; i++){ // i iterates over P's columns / M's columns
                sum += M[row*width+i] * N[i*width+j];
            }
            P[row*width+j] = sum;
        }
    }
}

__global__ void matmul_column_kernel(float *M, float *N, float *P, int width){
    /*
     * each thread compute one column of P
     * both block and grid size are 1D
     * i-th column of P involves the i-th column of N and all rows of M
     */
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    if(col < width){
        for(int j=0; j<width; j++){ // j iterates over M's rows
            float sum = 0;
            for(int i=0; i<width; i++){ // i iterates over P's rows / N's rows
                sum += M[j*width+i] * N[i*width+col];
            }
            P[j*width+col] = sum;
        }
    }
}

extern "C" void matmul_row(float *M, float *N, float *P, int width){
    float *d_M, *d_N, *d_P;
    int size = width * width * sizeof(float);

    hipMalloc(&d_M, size);
    hipMalloc(&d_N, size);
    hipMalloc(&d_P, size);

    hipMemcpy(d_M, M, size, hipMemcpyHostToDevice);
    hipMemcpy(d_N, N, size, hipMemcpyHostToDevice);

    matmul_row_kernel<<<(width+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE>>>(d_M, d_N, d_P, width);

    hipMemcpy(P, d_P, size, hipMemcpyDeviceToHost);

    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);
}

extern "C" void matmul_column(float *M, float *N, float *P, int width){
    float *d_M, *d_N, *d_P;
    int size = width * width * sizeof(float);

    hipMalloc(&d_M, size);
    hipMalloc(&d_N, size);
    hipMalloc(&d_P, size);

    hipMemcpy(d_M, M, size, hipMemcpyHostToDevice);
    hipMemcpy(d_N, N, size, hipMemcpyHostToDevice);

    matmul_column_kernel<<<(width+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE>>>(d_M, d_N, d_P, width);

    hipMemcpy(P, d_P, size, hipMemcpyDeviceToHost);

    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);
}

#include <hip/hip_runtime.h>
__global__ void matmul_row_kernel(float *M, float *N, float *P, int width){
    /*
     * each thread compute one row of P
     * both block and grid size are 1D
     * i-th row of P involves the i-th row of M and all columns of N
     */
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    if(row < width){
        int row_offset = row * width;
        for(int j=0; j<width; j++){
            P[row_offset+j] = 0;
        }
        for(int i=0; i<width; i++){ // i iterates M's columns
            for(int j=0; j<width; j++) { // j iterates over P's columns / N's columns
                P[row_offset+j] += M[row_offset+i] * N[i*width+j];
            }
        }
    }
}

extern "C" void matmul_row(float *M, float *N, float *P, int width){
    float *d_M, *d_N, *d_P;
    int size = width * width * sizeof(float);

    hipMalloc(&d_M, size);
    hipMalloc(&d_N, size);
    hipMalloc(&d_P, size);

    hipMemcpy(d_M, M, size, hipMemcpyHostToDevice);
    hipMemcpy(d_N, N, size, hipMemcpyHostToDevice);

    matmul_row_kernel<<<width, width>>>(d_M, d_N, d_P, width);

    hipMemcpy(P, d_P, size, hipMemcpyDeviceToHost);

    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);
}